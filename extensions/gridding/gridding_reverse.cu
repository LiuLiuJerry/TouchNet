#include "hip/hip_runtime.h"
/*
 * @Author: Haozhe Xie
 * @Date:   2019-11-21 16:42:18
 * @Last Modified by:   Haozhe Xie
 * @Last Modified time: 2020-06-17 15:00:21
 * @Email:  cshzxie@gmail.com
 */

#include <bits/stdc++.h>
#include <torch/extension.h>
#include <cstdio>
#include <cstdlib>

#define CUDA_NUM_THREADS 512
#define EPS 1e-6

// Computer the number of threads needed in GPU
inline int get_n_threads(int n) {
  const int pow_2 = std::log(static_cast<float>(n)) / std::log(2.0);
  return max(min(1 << pow_2, CUDA_NUM_THREADS), 1);
}

__device__ int compute_index(int offset_x,
                             int offset_y,
                             int offset_z,
                             int scale) {
  return offset_x * scale * scale + offset_y * scale + offset_z;
}

__global__ void gridding_reverse_kernel(int scale,
                                        int n_pts,
                                        const float *__restrict__ grid,
                                        float *__restrict__ ptcloud) {
  int batch_index = blockIdx.x;
  int index       = threadIdx.x;
  int stride      = blockDim.x;

  ptcloud += batch_index * n_pts * 3;
  grid += batch_index * n_pts;

  for (int j = index; j < n_pts; j += stride) { //第j个网格顶点对应的网格坐标
    int sqr_scale = scale * scale;
    int x_offset  = j / sqr_scale;
    int y_offset  = j % sqr_scale / scale;
    int z_offset  = j % sqr_scale % scale;
    if (x_offset == 0 || y_offset == 0 || z_offset == 0) {
      continue;
    }

    // assert j == compute_index(x_offset, y_offset, z_offset, scale)
    // 每个网格顶点的权重只有一维
    float weights[8] = {
      grid[compute_index(x_offset - 1, y_offset - 1, z_offset - 1, scale)],
      grid[compute_index(x_offset - 1, y_offset - 1, z_offset, scale)],
      grid[compute_index(x_offset - 1, y_offset, z_offset - 1, scale)],
      grid[compute_index(x_offset - 1, y_offset, z_offset, scale)],
      grid[compute_index(x_offset, y_offset - 1, z_offset - 1, scale)],
      grid[compute_index(x_offset, y_offset - 1, z_offset, scale)],
      grid[compute_index(x_offset, y_offset, z_offset - 1, scale)],
      grid[j]}; //取权重

    float weights_sum = 0;
    for (size_t i = 0; i < 8; ++i) {
      weights_sum += weights[i];
    }
    if (weights_sum < EPS) {
      continue;
    }
    for (size_t i = 0; i < 8; ++i) {
      weights[i] /= weights_sum;
    } //权重归一化

    x_offset -= scale / 2;
    y_offset -= scale / 2;
    z_offset -= scale / 2;

    // clang-format off  每个网格点算一个xyz坐标
    ptcloud[j * 3 + 0] = weights[0] * (x_offset - 1) +
                         weights[1] * (x_offset - 1) +
                         weights[2] * (x_offset - 1) +
                         weights[3] * (x_offset - 1) +
                         weights[4] * x_offset +
                         weights[5] * x_offset +
                         weights[6] * x_offset +
                         weights[7] * x_offset;
    ptcloud[j * 3 + 1] = weights[0] * (y_offset - 1) +
                         weights[1] * (y_offset - 1) +
                         weights[2] * y_offset +
                         weights[3] * y_offset +
                         weights[4] * (y_offset - 1) +
                         weights[5] * (y_offset - 1) +
                         weights[6] * y_offset +
                         weights[7] * y_offset;
    ptcloud[j * 3 + 2] = weights[0] * (z_offset - 1) +
                         weights[1] * z_offset +
                         weights[2] * (z_offset - 1) +
                         weights[3] * z_offset +
                         weights[4] * (z_offset - 1) +
                         weights[5] * z_offset +
                         weights[6] * (z_offset - 1) +
                         weights[7] * z_offset;
    // clang-format on
  }
}

torch::Tensor gridding_reverse_cuda_forward(int scale,
                                            torch::Tensor grid,
                                            hipStream_t stream) {
  int batch_size = grid.size(0);
  int n_pts      = scale * scale * scale; //每个网格产生一个点云顶点

  torch::Tensor ptcloud =
    torch::zeros({batch_size, n_pts, 3}, torch::CUDA(torch::kFloat));

  gridding_reverse_kernel<<<batch_size, get_n_threads(n_pts), 0, stream>>>(
    scale, n_pts, grid.data_ptr<float>(), ptcloud.data_ptr<float>());

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in gridding_cuda_forward: %s\n", hipGetErrorString(err));
  }
  return ptcloud;
}

//求梯度，将点云的梯度分散到每个网格上
__global__ void gridding_reverse_grad_kernel(
  int scale,
  int n_pts,
  const float *__restrict__ ptcloud,
  const float *__restrict__ grid,
  const float *__restrict__ grad_ptcloud,
  float *__restrict__ grad_grid) {
  //grid: 网格上的权重
  //ptcloud: 点云三维坐标(通过网格生成)
  //grad_ptcloud: 每个点云获得的梯度
  int batch_index = blockIdx.x;
  int index       = threadIdx.x;
  int stride      = blockDim.x;

  ptcloud += batch_index * n_pts * 3;
  grid += batch_index * n_pts;
  grad_ptcloud += batch_index * n_pts * 3;
  grad_grid += batch_index * n_pts;
  //取第j个顶点所在的网格坐标
  for (int j = index; j < n_pts; j += stride) {
    int sqr_scale = scale * scale;
    int x_offset  = j / sqr_scale;
    int y_offset  = j % sqr_scale / scale;
    int z_offset  = j % sqr_scale % scale;
    if (x_offset == 0 || y_offset == 0 || z_offset == 0) {
      continue;
    }
    //对应网格的8个顶点
    int gvtx_indexes[8] = {
      compute_index(x_offset - 1, y_offset - 1, z_offset - 1, scale),
      compute_index(x_offset - 1, y_offset - 1, z_offset, scale),
      compute_index(x_offset - 1, y_offset, z_offset - 1, scale),
      compute_index(x_offset - 1, y_offset, z_offset, scale),
      compute_index(x_offset, y_offset - 1, z_offset - 1, scale),
      compute_index(x_offset, y_offset - 1, z_offset, scale),
      compute_index(x_offset, y_offset, z_offset - 1, scale),
      j};
    float weights[8] = {grid[gvtx_indexes[0]], grid[gvtx_indexes[1]],
                        grid[gvtx_indexes[2]], grid[gvtx_indexes[3]],
                        grid[gvtx_indexes[4]], grid[gvtx_indexes[5]],
                        grid[gvtx_indexes[6]], grid[gvtx_indexes[7]]};

    float weights_sum = 0;
    for (size_t i = 0; i < 8; ++i) {
      weights_sum += weights[i];
    }

    if (weights_sum < EPS) {
      continue;
    } 
    //权重标准化
    for (size_t i = 0; i < 8; ++i) {
      weights[i] /= weights_sum;
    }

    x_offset -= scale / 2;
    y_offset -= scale / 2;
    z_offset -= scale / 2;

    // clang-format off  //每个顶点的梯度加上grad_ptcloud造成的影响
    // 每个点在每个维度给到的梯度从该点指向对应网格点，大小和点云在该维度的梯度成正比
    atomicAdd(&(grad_grid[gvtx_indexes[0]]),
                grad_ptcloud[j * 3 + 0] * ((x_offset - 1) - ptcloud[j * 3 + 0]) / weights_sum +
                grad_ptcloud[j * 3 + 1] * ((y_offset - 1) - ptcloud[j * 3 + 1]) / weights_sum +
                grad_ptcloud[j * 3 + 2] * ((z_offset - 1) - ptcloud[j * 3 + 2]) / weights_sum);
    atomicAdd(&(grad_grid[gvtx_indexes[1]]),
                grad_ptcloud[j * 3 + 0] * ((x_offset - 1) - ptcloud[j * 3 + 0]) / weights_sum +
                grad_ptcloud[j * 3 + 1] * ((y_offset - 1) - ptcloud[j * 3 + 1]) / weights_sum +
                grad_ptcloud[j * 3 + 2] * (z_offset - ptcloud[j * 3 + 2]) / weights_sum);
    atomicAdd(&(grad_grid[gvtx_indexes[2]]),
                grad_ptcloud[j * 3 + 0] * ((x_offset - 1) - ptcloud[j * 3 + 0]) / weights_sum +
                grad_ptcloud[j * 3 + 1] * (y_offset - ptcloud[j * 3 + 1]) / weights_sum +
                grad_ptcloud[j * 3 + 2] * ((z_offset - 1) - ptcloud[j * 3 + 2]) / weights_sum);
    atomicAdd(&(grad_grid[gvtx_indexes[3]]),
                grad_ptcloud[j * 3 + 0] * ((x_offset - 1) - ptcloud[j * 3 + 0]) / weights_sum +
                grad_ptcloud[j * 3 + 1] * (y_offset - ptcloud[j * 3 + 1]) / weights_sum +
                grad_ptcloud[j * 3 + 2] * (z_offset - ptcloud[j * 3 + 2]) / weights_sum);
    atomicAdd(&(grad_grid[gvtx_indexes[4]]),
                grad_ptcloud[j * 3 + 0] * (x_offset - ptcloud[j * 3 + 0]) / weights_sum +
                grad_ptcloud[j * 3 + 1] * ((y_offset - 1) - ptcloud[j * 3 + 1]) / weights_sum +
                grad_ptcloud[j * 3 + 2] * ((z_offset - 1) - ptcloud[j * 3 + 2]) / weights_sum);
    atomicAdd(&(grad_grid[gvtx_indexes[5]]),
                grad_ptcloud[j * 3 + 0] * (x_offset - ptcloud[j * 3 + 0]) / weights_sum +
                grad_ptcloud[j * 3 + 1] * ((y_offset - 1) - ptcloud[j * 3 + 1]) / weights_sum +
                grad_ptcloud[j * 3 + 2] * (z_offset - ptcloud[j * 3 + 2]) / weights_sum);
    atomicAdd(&(grad_grid[gvtx_indexes[6]]),
                grad_ptcloud[j * 3 + 0] * (x_offset - ptcloud[j * 3 + 0]) / weights_sum +
                grad_ptcloud[j * 3 + 1] * (y_offset - ptcloud[j * 3 + 1]) / weights_sum +
                grad_ptcloud[j * 3 + 2] * ((z_offset - 1) - ptcloud[j * 3 + 2]) / weights_sum);
    atomicAdd(&(grad_grid[gvtx_indexes[7]]),
                grad_ptcloud[j * 3 + 0] * (x_offset - ptcloud[j * 3 + 0]) / weights_sum +
                grad_ptcloud[j * 3 + 1] * (y_offset - ptcloud[j * 3 + 1]) / weights_sum +
                grad_ptcloud[j * 3 + 2] * (z_offset - ptcloud[j * 3 + 2]) / weights_sum);
    // clang-format on
  }
}

torch::Tensor gridding_reverse_cuda_backward(torch::Tensor ptcloud,
                                             torch::Tensor grid,
                                             torch::Tensor grad_ptcloud,
                                             hipStream_t stream) {
  int batch_size = ptcloud.size(0);
  int n_pts      = ptcloud.size(1);
  int scale      = static_cast<int>(std::cbrt(n_pts));

  torch::Tensor grad_grid =
    torch::zeros({batch_size, n_pts}, torch::CUDA(torch::kFloat));

  gridding_reverse_grad_kernel<<<batch_size, get_n_threads(n_pts), 0, stream>>>(
    scale, n_pts, ptcloud.data_ptr<float>(), grid.data_ptr<float>(),
    grad_ptcloud.data_ptr<float>(), grad_grid.data_ptr<float>());

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in gridding_cuda_forward: %s\n", hipGetErrorString(err));
  }
  return grad_grid;
}